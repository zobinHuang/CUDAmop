#include "hip/hip_runtime.h"
/*!
 * \file    vector_addition.cu
 * \brief   Operator for basic version of vector addition
 * \author  Zhuobin Huang
 * \date    July 25, 2022
 */

#include <iostream>
#include <vector>
#include <vector_addition.cuh>

/*!
 * \brief [CUDA Kernel] Conduct Vector Adding (a+b=c)
 * \param vector_a  source vector
 * \param vector_b  source vector
 * \param vector_c  destination vector
 * \param d         dimension of vectors
 */
__global__ void vectorAdd(
    const int *__restrict vector_a, 
    const int *__restrict vector_b, 
    int *__restrict vector_c, 
    int d){
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < d){
    vector_c[tid] = vector_a[tid] + vector_b[tid];
  }
}